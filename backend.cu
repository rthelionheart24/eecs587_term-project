#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "include/backend.h"
#include "include/data.h"

#define THREADS_PER_BLOCK 1024

BatchedGPUTask::BatchedGPUTask(QCircuit *qc, unsigned int num_shots) {
  this->circuit = qc;
  this->num_shots = num_shots;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  if (deviceCount == 0) {
    printf("No CUDA devices found\n");
    exit(EXIT_FAILURE);
  }

  hipError_t err = hipSetDevice(0);

  if (err != hipSuccess) {
    printf("Error setting CUDA device - %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  size_t free_mem, total_mem;

  hipError_t cuda_status = hipMemGetInfo(&free_mem, &total_mem);
  if (cuda_status != hipSuccess) {
    printf("Error getting CUDA memory info - %s\n",
           hipGetErrorString(cuda_status));
    exit(EXIT_FAILURE);
  }

  printf("Free memory: %lu\n", free_mem);
  printf("Total memory: %lu\n", total_mem);
}

BatchedGPUTask::~BatchedGPUTask() {
  hipError_t err = hipDeviceReset();

  if (err != hipSuccess) {
    printf("Error resetting CUDA device - %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

 __global__ void run(QCircuit *circuit) {
  return;
}

 void BatchedGPUTask::runWrapper() {

  printf("Running Simulation task\n");

  float *d_qubits;
  float *d_gates;
  float *d_bits;

  // Allocate memory on GPU
  hipError_t status;

  status =
      hipMalloc((void **)&d_qubits, sizeof(float) * this->circuit->num_qubits);

  if (status != hipSuccess) {
    printf("Error allocating memory for qubits on GPU - %s\n",
           hipGetErrorString(status));
    exit(EXIT_FAILURE);
  } else {
    printf("Allocated memory for qubits on GPU\n");
  }

  status =
      hipMalloc((void **)&d_bits, sizeof(float) * this->circuit->num_bits);

  if (status != hipSuccess) {
    printf("Error allocating memory for classical bits on GPU - %s\n",
           hipGetErrorString(status));
    exit(EXIT_FAILURE);
  } else {
    printf("Allocated memory for classical bits on GPU\n");
  }

  status =
      hipMalloc((void **)&d_gates, sizeof(float) * this->circuit->num_gates);

  if (status != hipSuccess) {
    printf("Error allocating memory for gates on GPU - %s\n",
           hipGetErrorString(status));
    exit(EXIT_FAILURE);
  } else {
    printf("Allocated memory for gates on GPU\n");
  }

  // Copy circuit to GPU
  status = hipMemcpy(d_qubits, this->circuit->qubits.data(),
                      sizeof(float) * this->circuit->num_qubits,
                      hipMemcpyHostToDevice);
  status = hipMemcpy(d_bits, this->circuit->bits.data(),
                      sizeof(float) * this->circuit->num_bits,
                      hipMemcpyHostToDevice);
  status = hipMemcpy(d_gates, this->circuit->gates.data(),
                      sizeof(float) * this->circuit->num_gates,
                      hipMemcpyHostToDevice);

  this->qubits_start_idx = 0;
  this->bits_start_idx = this->circuit->num_qubits;
  this->gates_start_idx = this->circuit->num_qubits + this->circuit->num_bits;

  if (status != hipSuccess) {
    printf("Error copying circuit to GPU - %s\n", hipGetErrorString(status));
    exit(EXIT_FAILURE);
  } else {
    printf("Copied circuit to GPU\n");
  }

  // Define grid and block dimensions
  dim3 gridDim;
  gridDim.x = 1;
  unsigned int size_circuit = this->circuit->num_qubits +
                              this->circuit->num_bits +
                              this->circuit->num_gates;

  gridDim.x = size_circuit / THREADS_PER_BLOCK;

  if (size_circuit % THREADS_PER_BLOCK != 0) {
    gridDim.x++;
  }

  printf("Grid dimensions: %d\n", gridDim.x);
  printf("Block dimensions: %d\n", THREADS_PER_BLOCK);

  // Run circuit on GPU

  run<<<gridDim, THREADS_PER_BLOCK>>>(this->circuit);

  hipDeviceSynchronize();

  // Copy results back to CPU
  status = hipMemcpy(this->circuit->qubits.data(), d_qubits,
                      sizeof(float) * this->circuit->num_qubits,
                      hipMemcpyDeviceToHost);
  status = hipMemcpy(this->circuit->bits.data(), d_bits, sizeof(float) * this->circuit->num_bits,
                      hipMemcpyDeviceToHost);
  status = hipMemcpy(this->circuit->gates.data(), d_gates,
                      sizeof(float) * this->circuit->num_gates , hipMemcpyDeviceToHost);

  if (status != hipSuccess) {
    printf("Error copying results back to CPU - %s\n",
           hipGetErrorString(status));
    exit(EXIT_FAILURE);
  } else {
    printf("Copied results back to CPU\n");
  }

  hipFree(d_qubits);
  hipFree(d_bits);
  hipFree(d_gates);
 }
